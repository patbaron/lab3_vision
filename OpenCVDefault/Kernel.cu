#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

int iDivUp(const int a, const int b) { return (a % b != 0) ? (a / b + 1) : (a / b); };

__global__ static void KernelRect(unsigned char *imgdst,long *X,long *Y, int imgWidth, int imgHeight)
{
	unsigned long index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum = 0;

	//sommation Y
	if(index < imgHeight)
	{
		for(int i = 0; i < imgWidth;i++)
			sum = sum + imgdst[index * imgWidth + i];
		Y[index] = sum;
	}

	sum = 0;

	//sommation X
	if(index < imgWidth)
	{
		for(int i = 0; i < imgHeight; i++)
			sum = sum + imgdst[index + imgWidth * i];
		X[index] = sum;
	}
}

__global__ static void KernelSobel(unsigned char *imgscr,unsigned char *imgscr2, unsigned char *imgdst,int imgWidth,int imgHeight,unsigned char K)
{
	unsigned long index;
	unsigned char pValue;

	index = threadIdx.x + blockIdx.x * blockDim.x;
	pValue = abs(imgscr[index] - imgscr2[index]);
	
	if(pValue < K)
		imgdst[index] = 0;
	else
		imgdst[index] =255;
	
}

extern "C" void Mykernelfunc(unsigned char *c_imgsrc, unsigned char *c_imgsrc2, unsigned char *c_imgdst, long *X, long *Y, int imageWidth, int imageHeight, unsigned long imagesize, unsigned char K)
{
	float tempsex = 0;
	
	// Defining gpu variables
	unsigned char *gpu_imgsrc = 0;
	unsigned char *gpu_imgsrc2 = 0;
	unsigned char *gpu_imgdst = 0;
	long *gpu_X = 0;
	long *gpu_Y = 0;

	// Defining size of memory allocations
	dim3 dimBlock = 16;
	dim3 dimGrid = imagesize / dimBlock.x;
	dim3 dimGridsum = 0;

	// Grabbing the highest value of both imageheight and imagewidth
	if(imageHeight > imageWidth)
		dimGridsum = iDivUp(imageHeight, dimBlock.x);
	else
		dimGridsum = iDivUp(imageWidth, dimBlock.x);

	// Finding the memory size of the image
	size_t memsize = imagesize * sizeof(unsigned char);

	//Allocating memory
	hipMalloc((void**)&gpu_imgsrc, memsize);
	hipMemcpy(gpu_imgsrc, c_imgsrc, memsize, hipMemcpyHostToDevice);
	hipMalloc((void**)&gpu_imgsrc2, memsize);
	hipMemcpy(gpu_imgsrc2, c_imgsrc2, memsize, hipMemcpyHostToDevice);
	hipMalloc((void**)&gpu_imgdst, memsize);
	hipMalloc((void**)&gpu_X,imageWidth * sizeof(long));
	hipMemcpy(gpu_X, X, imageWidth * sizeof(long), hipMemcpyHostToDevice);
	hipMalloc((void**)&gpu_Y,imageHeight * sizeof(long));
	hipMemcpy(gpu_Y, Y, imageHeight * sizeof(long), hipMemcpyHostToDevice);
	
	// Launching kernel functions
	KernelSobel<<<dimGrid.x, dimBlock.x>>>(gpu_imgsrc, gpu_imgsrc2, gpu_imgdst, imageWidth, imageHeight, K);
	hipDeviceSynchronize();

	KernelRect<<<dimGridsum.x,dimBlock.x>>>(gpu_imgdst, gpu_X, gpu_Y, imageWidth, imageHeight);
	hipDeviceSynchronize();

	// Grabbing data from gpu variables
	hipMemcpy(c_imgdst,gpu_imgdst, memsize, hipMemcpyDeviceToHost);
	hipMemcpy(X,gpu_X,imageWidth * sizeof(long), hipMemcpyDeviceToHost);
	hipMemcpy(Y,gpu_Y,imageHeight * sizeof(long), hipMemcpyDeviceToHost);

	// Freeing memory
	hipFree(gpu_imgdst);
	hipFree(gpu_imgsrc);
	hipFree(gpu_imgsrc2);
	hipFree(gpu_X);
	hipFree(gpu_Y);
}
